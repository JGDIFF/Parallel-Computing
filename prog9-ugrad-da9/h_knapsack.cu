#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "helpers.h"
#include "h_knapsack.h"
#include "wrappers.h"

//for printing out the entire best array
#define DEBUG 0

//prototype for function local to this file
static void knapsackOnCPU(int * result, int * weights, int * values, int numObjs, int capacity); 

/*  h_knapsack
    This function prepares and invokes a kernel to solve the 0-1 knapsack problem
    on the CPU. The input to the knapsack problem is a set of objects and a 
    knapsack capacity.  Each object has a weight and a value. The solution chooses a subset 
    of the objects that maximums the overall value while not exceeding the capacity.
    Inputs:
    result - points to an array to hold the knapsack result
    weights - points to an array that holds the weights of the objects
    values - points to an array that holds the values of the objects
    numObjs - number of objects (size of values and weights arrays)
    capacity - the capacity of the knapsack
*/
float h_knapsack(int * result, int * weights, int * values, int numObjs, int capacity) 
{
    hipEvent_t start_cpu, stop_cpu;
    float cpuMsecTime = -1;

    //CUERR is a macro defined in helpers.h that checks for a CUDA error
    //Use CUDA functions to do the timing 
    //Create event objects
    hipEventCreate(&start_cpu);                       CUERR
    hipEventCreate(&stop_cpu);                        CUERR
    //record the starting time
    hipEventRecord(start_cpu);                        CUERR
    
    //call function that does the actual work
    knapsackOnCPU(result, weights, values, numObjs, capacity);
   
    //record the ending time and wait for event to complete
    hipEventRecord(stop_cpu);                              CUERR
    hipEventSynchronize(stop_cpu);                         CUERR

    //calculate the elapsed time between the two events 
    hipEventElapsedTime(&cpuMsecTime, start_cpu, stop_cpu); CUERR 
    return cpuMsecTime;
}

/*  knapsackOnCPU
    This function performs the 0-1 knapsack on the CPU.  
    Inputs:
    result - points to an array to hold the knapsack result
    weights - points to an array that holds the weights of the objects
    values - points to an array that holds the values of the objects
    numObjs - number of objects (size of values and weights arrays)
    capacity - the capacity of the knapsack
*/
void knapsackOnCPU(int * result, int * weights, int * values, int numObjs, int capacity) 
{
    int i, j; 
    int nRows = numObjs + 1;
    int nCols = capacity + 1;

    //allocate space for the best array`
    int bestArraySz = (nRows * nCols);
    int * best = (int *) Malloc(sizeof(int) * bestArraySz);

    //initialize it 
    memset(best, 0, sizeof(int) * bestArraySz);

    //solve the knapsack problem using dynamic programming
    for (i = 1; i < nRows; i++) 
    {
        for (j = 0; j < nCols; j++)
        {
            if (weights[i - 1] <= j)
            {
                best[i * nCols + j] = MAX(best[(i - 1) * nCols + j],
                                          best[(i - 1) * nCols  + j - weights[i - 1]] + values[i - 1]); 
            } else
            {
                best[i * nCols + j] = best[(i - 1) * nCols + j];
            }
        }
    }
    if (DEBUG)
    {
        for (i = 0; i < nRows; i++) 
        {
            for (j = 0; j < nCols; j++)
            {
                printf("%4d ", best[i * nCols + j]);
            }
            printf("\n");
        }
    }
    
    //copy last row into result array
    for (i = 0; i < nCols; i++) result[i] = best[numObjs * nCols + i];
}
