#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "helpers.h"
#include "d_knapsack.h"

//prototypes for kernels in this file
__global__ 
void d_knapsackNaiveKernel(int * d_best, int * d_weight, int * d_values, int numObjs,
                           int capacity);

__global__ 
void d_knapsackOptKernel(int * d_best, int * d_weight, int * d_values, int numObjs,
                           int capacity);

/*  d_knapsack
    This function prepares and invokes a kernel to solve the 0-1 knapsack problem
    on the GPU. The input to the knapsack problem is a set of objects and a 
    knapsack capacity.  Each object has a weight and a value. The solution chooses a subset 
    of the objects that maximums the overall value while not exceeding the capacity.
    Inputs:
    result - points to an array to hold the knapsack result
    weights - points to an array that holds the weights of the objects
    values - points to an array that holds the values of the objects
    numObjs - number of objects (size of values and weights arrays)
    capacity - the capacity of the knapsack
    blkDim - the number of threads in the block of threads used to solve the problem
    which - indicates which kernel to use to solve the problem (NAIVE, OPT)
*/
float d_knapsack(int * result, int * weights, int * values, int numObjs, 
                 int capacity, int blkDim, int which)
{
    int * d_best, * d_weights, * d_values;  //pointers to arrays for GPU
   
    //CUERR is a macro in helpers.h that checks for a Cuda error 
    //Begin the timing (macro in helpers.h) 
    TIMERSTART(gpuTime)

    //Allocate space in GPU memory for weights array 
    hipMalloc((void **)&d_weights, sizeof(int) * numObjs);             CUERR
    //Copy weights from CPU memory to GPU memory
    hipMemcpy(d_weights, weights, sizeof(int) * numObjs, H2D);         CUERR

    //Allocate space in GPU memory for values array 
    hipMalloc((void **)&d_values, sizeof(int) * numObjs);              CUERR
    //Copy values from CPU memory to GPU memory
    hipMemcpy(d_values, values, sizeof(int) * numObjs, H2D);           CUERR

    //Launch the appropriate kernel
    if (which == NAIVE)
    {
        //Allocate space in GPU memory for best array
        int bestSz = (numObjs + 1) * (capacity + 1);
        hipMalloc((void **)&d_best, sizeof(int) * bestSz);             CUERR
        //set the best matrix to 0
        hipMemset((void *)d_best, 0, bestSz * sizeof(int));            CUERR
        //define the block and the grid and launch the naive kernel
        dim3 block(blkDim, 1, 1);
        dim3 grid(1, 1, 1);
        d_knapsackNaiveKernel<<<grid, block>>>(d_best, d_weights, d_values,
                                               numObjs, capacity);     CUERR
        //copy last row of d_best array into result
        hipMemcpy(result, 
                   &d_best[numObjs * (capacity + 1)], sizeof(int) * (capacity + 1),
                   D2H);                                               CUERR
    } else if (which == OPT)
    {
        //TO DO
        //Provide the code that is missing to execute the optimized kernel
        //1) Allocate space for the best array. 
        //   Note best array size is not the same naive best array.
        //2) Set best array elements to 0
        //3) Define block and grid
        //4) Launch the kernel
        //5) Copy best matrix into CPU result array

	//Allocate space in GPU memory for best array
        int bestSz = (2) * (capacity + 1);
        hipMalloc((void **)&d_best, sizeof(int) * bestSz);             CUERR
        //set the best matrix to 0
        hipMemset((void *)d_best, 0, bestSz * sizeof(int));            CUERR
        //define the block and the grid and launch the naive kernel
        dim3 block(blkDim, 1, 1);
        dim3 grid(1, 1, 1);
        d_knapsackOptKernel<<<grid, block>>>(d_best, d_weights, d_values,
                                               numObjs, capacity);     CUERR
        //copy last row of d_best array into result
        hipMemcpy(result, 
                   &d_best[0], sizeof(int) * (capacity + 1),
                   D2H);                                               CUERR

    }
    //free dynamically  allocated memory
    hipFree(d_best);                                                 CUERR
    hipFree(d_values);                                               CUERR
    hipFree(d_weights);                                              CUERR

    //stop the timer
    TIMERSTOP(gpuTime)
    return TIMEELAPSED(gpuTime)
}

/*  
    d_knapsackNaiveKernel
    This kernel solves the knapsack problem using a naive kernel.
    Inputs:
    best - pointer to the array in which the result is stored
    weights - points to an array that holds the weights of the objects
    values - points to an array that holds the values of the objects
    numObjs - number of objects (size of values and weights arrays)
    capacity - the capacity of the knapsack
*/

__global__
void d_knapsackNaiveKernel(int * best, int * weights, int * values, 
                           int numObjs, int capacity)
{
    //TO DO

    //You should base this implementation on the CPU version in h_knapsack.cu. 
    //1) Do not allocate and initalize the best array here. That was done before
    //   the kernel launch.
    //2) Each thread will loop through the number of rows (just like CPU version)
    //3) In the inner, loop cyclic partitioning will be used to divide up the 
    //   the row elements (the columns) among the threads.  For example, 
    //   thread 0 will calculate the 0th element, the blockDim.x element, 
    //   the 2*blockDim.x element, etc in the ith row of best. 
    //   (Note that best is a 1D array holding 2D data.)
    //   Thus, all threads of a block will cooperate in producing one row (i) 
    //   of results.
    //4) Block synchronization is needed so the threads in a block won't continue
    //   onto next row until all threads are finished with the current row.
    int i, j;
    int nRows = numObjs + 1;
    int nCols = capacity + 1;
    
    for (i = 1; i < nRows; i++) 
    {
	for (j = threadIdx.x; j < nCols; j += blockDim.x) 
	{
	    if (weights[i - 1] <= j)
	    {
	    	best[i * nCols + j] = MAX(best[(i - 1) * nCols + j],
                                          best[(i - 1) * nCols  + j - weights[i - 1]] + values[i - 1]);
            } else {
	        best[i * nCols + j] = best[(i - 1) * nCols + j];
	    }
	}
	__syncthreads();
    }
}

/*  
    d_knapsackOptKernel
    This kernel solves the knapsack problem using an optimized kernel.
    Inputs:
    best - pointer to the array in which the result is stored
    weights - points to an array that holds the weights of the objects
    values - points to an array that holds the values of the objects
    numObjs - number of objects (size of values and weights arrays)
    capacity - the capacity of the knapsack
*/
__global__ 
void d_knapsackOptKernel(int * best, int * weights, int * values, int numObjs,
                         int capacity)
{
    //TO DO

    //For this one, you'll start with the naive kernel code and improve it.
    //1) Note all of the accesses to weights[i-1] and values[i-1] that occur in
    //   in the inner most loop.  Those are all accesses to global memory.  You want
    //   to change those so that they are either accesses to the faster
    //   shared memory or to registers. (This is a really easy fix and one that
    //   programmers should always do whether for CPU code or GPU code.)
    //2) Use less global memory for the best array.  Note that the last row
    //   of the best array is what is copied into result. And each row is computed
    //   using the values in the previous row. Thus, this code can
    //   be implemented with just two rows in the best array.

    int i, j;
    int nRows = numObjs + 1;
    int nCols = capacity + 1;
    int srcI = 0;
    int dstI = 1;
    int tmp = 0; 
    
    for (i = 1; i < nRows; i++) 
    {
	for (j = threadIdx.x; j < nCols; j += blockDim.x) 
	{
	    if (weights[i - 1] <= j)
	    {
	    	best[dstI * nCols + j] = MAX(best[(srcI) * nCols + j],
                                          best[(srcI) * nCols  + j - weights[i - 1]] + values[i - 1]);
            } else {
	        best[dstI * nCols + j] = best[(srcI) * nCols + j];
	    }
	}
	tmp = srcI; 
	srcI = dstI;
	dstI = tmp;
	__syncthreads();
    }
}

