#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include "helpers.h"
#include "d_transpose.h"

//tile size for optimized kernel must be 4
#define OPTTILESZ 4

//prototypes for kernels in this file
__global__ 
void d_transposeNaiveKernel(float * d_result, float * d_input, int width);

__global__ 
void d_transposeTiledKernel(float * d_result, float * d_input, int width, int tileSz);

__global__ 
void d_transposeOptTiledKernel(float * d_result, float * d_input, int width);

/*  d_transpose
    This function prepares and invokes a kernel to perform
    a matrix transpose on the GPU. The matrices have been 
    linearized so each array is 1D and contains width * width elements.
    Inputs:
    result - points to a matrix to hold the transposed result
    input - points to the input matrix 
    width - width and height of the input and result matrices
    blkDim - dimensions of each block of threads to be launched
    tileSz - dimension of the size of a tile of elements to be handled
             by one thread (1 for NAIVE version, 4 for OPTTILED,
             TILED version can be 1, 2, 4, 8, or 16)
    which - indicates which kernel to use (NAIVE, TILED, OPTTILED)
*/
float d_transpose(float * result, float * input, int width, int blkDim, 
                  int tileSz, int which)
{
    float * d_result, * d_input;  //pointers to matrices for GPU
   
    //CUERR is a macro in helpers.h that checks for a Cuda error 
    //Begin the timing (macro in helpers.h) 
    TIMERSTART(gpuTime)
    //Allocate space in GPU memory for input matrix
    hipMalloc((void **)&d_input, sizeof(float) * width * width);            CUERR
    //Copy input from CPU memory to GPU memory
    hipMemcpy(d_input, input, sizeof(float) * width * width, H2D);          CUERR
    //Allocate space in GPU memory for result matrix
    hipMalloc((void **)&d_result, sizeof(float) * width * width);           CUERR

    //Launch the appropriate kernel
    if (which == NAIVE)
    {
        //Define the block and the grid and launch the naive kernel
        int grdDim = SDIV(width, blkDim); 
        dim3 block(blkDim, blkDim, 1);
        dim3 grid(grdDim, grdDim, 1);
        d_transposeNaiveKernel<<<grid, block>>>(d_result, d_input, width);   CUERR
    } else if (which == TILED)
    {
	int grdDim = SDIV(width, blkDim);
	dim3 block(blkDim, blkDim, 1);
	dim3 grid(grdDim, grdDim, 1);
	d_transposeTiledKernel<<<grid, block>>>(d_result, d_input, width, tileSz);
        //TO DO
        //Define the block and the grid and launch the tiled kernel
        //This is similar to above, but not exactly the same.
        //You'll need to use the width, blkDim, and tileSz parameters.
        //Be careful to not define a grid that is too big
    } else if (which == OPTTILED)
    {
        //TO DO
        //Define the block and the grid and launch the optimized tiled kernel
        //Be careful to not define a grid that is too big
    }
    
    //wait for threads to finish
    hipDeviceSynchronize();                                                  CUERR
    //copy result from GPU memory to CPU memory
    hipMemcpy(result, d_result, sizeof(float) * width * width, D2H);         CUERR

    //free dynamically  allocated memory
    hipFree(d_result);                                                       CUERR
    hipFree(d_input);                                                        CUERR

    //stop the timer
    TIMERSTOP(gpuTime)
    return TIMEELAPSED(gpuTime);
}

/*  
    d_transposeNaiveKernel
    This kernel performs a naive transpose of an input matrix 
    and stores the result in the d_result matrix.
    Each matrix is of size width by width and has been linearized.
    Each thread performs the transpose of element.  
    Inputs:
    d_result - pointer to the array in which the result is stored
    d_input - pointer to the array containing the input
    width - width and height of the matrices
*/
__global__ 
void d_transposeNaiveKernel(float * d_result, float * d_input, int width) 
{
    //Working :D 
    //TO DO
    //1) Use the blockIdx.y, blockDim.y and threadIdx.y
    //   to calculate the row of the input matrix
    int i = 0; //input array index
    int j = 0; //result array index 
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    //2) Use the blockIdx.x, blockDim.x and threadIdx.x
    //   to calculate the col of the input matrix
    int col = blockDim.x * blockIdx.x + threadIdx.x;


    //3) Flatten the row and col to determine an index into the input array 
    //i = row * gridDim.x + col;
    //4) Flatten the col and row to determine an index into the result array
    //j = row * gridDim.x + col;
    //5) Do the tranpose

    //Be careful to not access outside of the dimensions of the arrays.
    if (row < width && col < width) 
    {
      d_result[col * width + row] = d_input[row * width + col];
    }
}      

/*  
    d_transposeTiledKernel
    This kernel performs a tiled transpose of an input matrix 
    and stores the result in the d_result matrix.
    Each matrix is of size width by width and has been linearized.
    Each thread performs the transpose of tile by tile elements.  
    Inputs:
    d_result - pointer to the array in which the result is stored
    d_input - pointer to the array containing the input
    width - width and height of the matrices
*/
__global__ 
void d_transposeTiledKernel(float * d_result, float * d_input,
                            int width, int tileSz) 
{

    //TO DO
    //1) Use the blockIdx.y, blockDim.y, threadIdx.y, and tileSz
    //   to calculate the smallest row of a tile of the input matrix.
    int i = 0;
    int j = 0;
    int flatinput, flatresult = 0;
    int row = (blockDim.y * blockIdx.y + threadIdx.y) * tileSz;
    //2) Use the blockIdx.x, blockDim.x and threadIdx.x, and tileSz
    //   to calculate the smallest col of a tile of the input matrix.
    int col = (blockDim.x * blockIdx.x + threadIdx.x) * tileSz;
    //3) Loop through the tile of the input matrix. 
    for (i = 0; i < tileSz; i++) {
      for (j = 0; j < tileSz; j++) {
        flatinput = (row + i) * width + col + j;
        flatresult = (col + j) * width + row + i;
        if (row < width && col < width) {
          d_result[flatresult] = d_input[flatinput];
        }
      }
    }
    //4) In the loop, flatten the current row and current col to determine an index 
    //   into the input array.
    //5) In the loop, flatten the current col and current row to determine an index 
    //   into the result array.
    //6) In the loop, copy the input element into the result array.

    //Be careful to not access outside of the bounds of the
    //input and result matrices.
}      

/*
 * swap
 * Swap the contents of two floats in the device memory.
 * Inputs
 * fval1 - pointer to one of the floats
 * fval2 - pointer to the other float
 * Result
 * (*fval1) and (*fval2) values are swapped
*/
__device__
void swap(float * fval1, float * fval2)
{
   float tmp;
   tmp = (*fval1);
   (*fval1) = (*fval2);
   (*fval2) = tmp;
}


/*  
    d_transposeOptTiledKernel
    This kernel performs a optimized tiled transpose of an input matrix 
    and stores the result in the d_result matrix.
    Each matrix is of size width by width and has been linearized.
    Each thread performs the transpose of 16 elements.  
    Inputs:
    d_result - pointer to the array in which the result is stored
    d_input - pointer to the array containing the input
    width - width and height of the matrices
*/
__global__ 
void d_transposeOptTiledKernel(float * d_result, float * d_input, int width)
{
    float tile[OPTTILESZ][OPTTILESZ];

    int row = (blockDim.y * blockIdx.y + threadIdx.y) * OPTTILESZ;
    int col = (blockDim.x * blockIdx.x + threadIdx.x) * OPTTILESZ;
    int i = 0;
    int j = 0;

    for(int i = 0; i < OPTTILESZ; i++) {
        for (int j = 0; j < OPTTILESZ; j++) {
            flatinput = (row + i) * width + col + j;
            flatresult = (col + j) * width + row + i;
            if (row < width && col < width) {
                d_result[flatresult] = d_input[flatinput];
        }   
    }

    //TO DO
    //1) Use the blockIdx.y, blockDim.y, threadIdx.y, and OPTTILESZ
    //   to calculate the smallest row of a tile of the input matrix.
    //2) Use the blockIdx.x, blockDim.x and threadIdx.x, and OPTTILESZ
    //   to calculate the smallest col of a tile of the input matrix.


    //3) Loop through the tile of the input matrix, copying elements 
    //   from the input array into the tile array. Instead of copying 
    //   16 float values, copy 8 doubles (two doubles per row). You can do 
    //   this trick using double pointers. 

    
    //4) Do the transpose in the tile array (six swaps).
    //5) Use a loop to copy the elements in the tile array into the result 
    //   array.  Again, you'll use double pointers so that you end up 
    //   copying 8 doubles instead of 16 floats.

    //Of course like the previous kernels, you'll need to flatten the row and
    //column values.

    //Be careful to not access outside of the bounds of the
    //input and result matrices.

}      

